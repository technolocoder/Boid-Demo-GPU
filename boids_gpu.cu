#include "hip/hip_runtime.h"
#include <SDL2/SDL.h>
#include <random>
#include <iostream>

#define CUDA_GLOBAL_FUNC __global__
#define CUDA_DEVICE_FUNC __device__
#define CUDA_HOST_FUNC __host__
#define CUDA_CONSTANT __constant__

using namespace std;

CUDA_CONSTANT float _max_distance;
CUDA_CONSTANT int _boid_count, _window_width, _window_height;

struct vec2{
    float x,y;
};

CUDA_CONSTANT vec2 _mouse_pos;

struct boid{
    vec2 position, velocity;
};

CUDA_DEVICE_FUNC float get_distance(const vec2 &a ,const vec2 &b){
    float diffx = a.x-b.x;
    float diffy = a.y-b.y;
    return __fsqrt_rn(diffx*diffx+diffy*diffy);
}

CUDA_DEVICE_FUNC float get_magnitude(const vec2 &a){
    return __fsqrt_rn(a.x*a.x+a.y*a.y);
}

CUDA_DEVICE_FUNC vec2 normalize_vec(const vec2 &a){
    float magnitude = get_magnitude(a);
    return {a.x/magnitude,a.y/magnitude};
}

CUDA_GLOBAL_FUNC void compute_acc(boid *boids, vec2 *acc){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for(int i = index; i < _boid_count; i += stride){
        vec2 out {0.0f,0.0f}, norm_avg = normalize_vec(boids[i].velocity),pos_avg = boids[i].position;
        int count = 1;
        for(int j = 0; j < i; ++j){
            float dist = get_distance(boids[i].position,boids[j].position);
            if(dist < _max_distance){
                ++count;
                vec2 diff {boids[j].position.x-boids[i].position.x,boids[j].position.y-boids[i].position.y};
                vec2 direction = normalize_vec(diff);
                vec2 bdir = normalize_vec(boids[j].velocity);
                norm_avg.x += bdir.x;
                norm_avg.y += bdir.y;
                pos_avg.x += boids[j].position.x;
                pos_avg.y += boids[j].position.y;
                float force = max((30.0-dist)/1000,0.0);

                out.x -= direction.x * force;
                out.y -= direction.y * force;
            }
        }
        for(int j = i+1; j < _boid_count; ++j){
            float dist = get_distance(boids[i].position,boids[j].position);
            if(dist < _max_distance){
                ++count;
                vec2 diff {boids[j].position.x-boids[i].position.x,boids[j].position.y-boids[i].position.y};
                vec2 direction = normalize_vec(diff);
                vec2 bdir = normalize_vec(boids[j].velocity);
                norm_avg.x += bdir.x;
                norm_avg.y += bdir.y;
                pos_avg.x += boids[j].position.x;
                pos_avg.y += boids[j].position.y;
                float force = max((30.0-dist)/1000,0.0);

                out.x -= direction.x * force;
                out.y -= direction.y * force;
            }
        }
        norm_avg.x /= count;
        norm_avg.y /= count;

        pos_avg.x /= count;
        pos_avg.y /= count;

        vec2 diff = {norm_avg.x-boids[i].velocity.x,norm_avg.y-boids[i].velocity.y};
        vec2 norm = normalize_vec(diff);
        out.x += norm.x*0.0016;
        out.y += norm.y*0.0016;

        vec2 pos_diff = {pos_avg.x-boids[i].position.x,pos_avg.y-boids[i].position.y};
        vec2 pos_norm = normalize_vec(pos_diff);

        out.x += norm.x * 0.0016;
        out.y += norm.y * 0.0016;

        float mouse_dist = get_distance(boids[i].position,_mouse_pos);
        if(mouse_dist < _max_distance){
            vec2 mouse_diff = {boids[i].position.x-_mouse_pos.x,boids[i].position.y-_mouse_pos.y};;
            vec2 mouse_norm = normalize_vec(mouse_diff);

            float force = max((50.0-mouse_dist)/500,0.0);
            out.x += mouse_norm.x * force;
            out.y += mouse_norm.y * force;
        }

        if(boids[i].position.x < 80){
            out.x += max(80.0-boids[i].position.x,0.0)/1000.0;
        }else if(boids[i].position.x > _window_width-80){
            out.x -= max(80.0-(_window_width-boids[i].position.x),0.0)/1000.0;
        }   
        
        if(boids[i].position.y < 80){
            out.y += max(80.0-boids[i].position.y,0.0)/1000.0;
        }else if(boids[i].position.y > _window_height-80){
            out.y -= max(80.0-(_window_height-boids[i].position.y),0.0)/1000.0;
        }
        

        acc[i] = out;
    }
}

CUDA_GLOBAL_FUNC void update_boids(boid *boids ,const vec2 *acc){
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for(int i = index; i < _boid_count; i += stride){
        boids[i].velocity.x += acc[i].x;
        boids[i].velocity.y += acc[i].y;
        
        boids[i].position.x += boids[i].velocity.x;
        boids[i].position.y += boids[i].velocity.y;
    }
}

int main(){
    SDL_Init(SDL_INIT_VIDEO);
    
    SDL_DisplayMode display_mode;
    SDL_GetDesktopDisplayMode(0,&display_mode);
    int window_width = display_mode.w ,window_height = display_mode.h;

    SDL_Window *window = SDL_CreateWindow("Boids",SDL_WINDOWPOS_CENTERED,SDL_WINDOWPOS_UNDEFINED,window_width,window_height,SDL_WINDOW_SHOWN|SDL_WINDOW_FULLSCREEN);
    SDL_Renderer *renderer = SDL_CreateRenderer(window,-1,SDL_RENDERER_ACCELERATED);

    SDL_Event event;
    bool quit = false;

    const int fps = 60;
    const int frame_delay = 1000/fps;
    unsigned int current,reference=0;

    const int boid_count = 200;
    const float max_distance = 50.0f;

    hipMemcpyToSymbol(HIP_SYMBOL(_max_distance),&max_distance,sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(_boid_count),&boid_count,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(_window_width),&window_width,sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(_window_height),&window_height,sizeof(int));

    boid *boids;
    hipMallocManaged(&boids,boid_count*sizeof(boid));

    vec2 *acc;
    hipMallocManaged(&acc,boid_count*sizeof(vec2));

    random_device rd;
    mt19937_64 engine(rd());
    uniform_int_distribution<int> dist_x(0,window_width), dist_y(0,window_height);
    uniform_real_distribution<float> dist_vel(-1.0f,1.0f);
    for(int i = 0; i < boid_count; ++i){
        boids[i].position = {(float)dist_x(engine),(float)dist_y(engine)};
        boids[i].velocity = {dist_vel(engine),dist_vel(engine)};
    }

    while(!quit){
        while(SDL_PollEvent(&event)){
            if(event.type == SDL_QUIT){
                quit = true;
                break;
            }else if(event.type == SDL_KEYDOWN){
                switch(event.key.keysym.sym){
                    case SDLK_ESCAPE:
                        quit = true;
                        break;
                }
            }else if(event.type == SDL_MOUSEMOTION){
                vec2 pos = {(float)event.motion.x,(float)event.motion.y};
                hipMemcpyToSymbol(HIP_SYMBOL(_mouse_pos),&pos,sizeof(vec2));
            }
        }
        current = SDL_GetTicks();
        if(current-reference > frame_delay){
            reference = current;

            SDL_SetRenderDrawColor(renderer,0,0,0,0xFF);
            SDL_RenderClear(renderer);

            SDL_SetRenderDrawColor(renderer,0xFF,0xFF,0xFF,0xFF);

            compute_acc<<<(boid_count+255)/256,256>>>(boids,acc);
            hipDeviceSynchronize();

            update_boids<<<(boid_count+255)/256,256>>>(boids,acc);
            hipDeviceSynchronize();
            SDL_Point points[boid_count];
            for(int i = 0; i < boid_count; ++i) points[i] = {(int)boids[i].position.x,(int)boids[i].position.y};
            SDL_RenderDrawPoints(renderer,points,boid_count);

            SDL_RenderPresent(renderer);           
        }
    }

    hipFree(boids);
    hipFree(acc);

    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();
    return 0;
}